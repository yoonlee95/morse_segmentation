
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

__global__ void transpose(float* A, float* B, int m, int n)
{
	__shared__ float sm[BLOCK_SIZE][BLOCK_SIZE];

	int tx = threadIdx.x; 	int ty = threadIdx.y;
	int bx = blockIdx.x; 	int by = blockIdx.y;

	int row = by * blockDim.y + ty;
	int col = bx * blockDim.x + tx;		

	if(row<m && col <n)
		sm[ty][tx] = A[row*n+col];
	__syncthreads();

	row = bx * blockDim.y + ty;
	col = by * blockDim.x + tx;

	if(row<n && col < m)
		B[row*m+col] = sm[tx][ty];
	__syncthreads();

	return;
}
