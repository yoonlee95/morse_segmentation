
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_DIM_X 32 
#define BLOCK_DIM_Y 16 

#define VECTOR_DIM 300
#define PARTITION_DIM 32

__global__ void
vectorAdd(float *A, const float *B,unsigned int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = B[i] - A[i];
    }
}


__global__ void 
L2_norm(float *g_idata, float *g_odata)
{
  __shared__ float sdata[256];

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = g_idata[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * A; 

  if(tid + blockDim.x < 300){   // blockDimx.x = 256
    A = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * A;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   g_odata[blockIdx.x] = sqrtf(sdata[0]);
  }

}
__global__ void 
Dot_product(float *A_input, float *B_input,float * output)
{
  __shared__ float sdata[256];

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = A_input[tid + VECTOR_DIM * blockIdx.x];
  float B = B_input[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * B; 

  if(tid + blockDim.x < 300){
    A = A_input[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    B = B_input[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * B;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   output[blockIdx.x] = (sdata[0]);
  }

}



__global__ void 
Normalize_vector(float *g_idata)
{
  __shared__ float sdata[256];
  __shared__ float norm;

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = g_idata[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * A; 

  if(tid + blockDim.x < 300){
    A = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * A;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   norm = sqrtf(sdata[0]);
  }
  __syncthreads();

  g_idata[tid + VECTOR_DIM * blockIdx.x] = g_idata[tid + VECTOR_DIM * blockIdx.x]/norm;

  if(tid + blockDim.x < 300){
    g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x] = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x]/ norm;
  }

}


__global__ void 
cos_distance(float *A, float *B_L2_NORM, int* Count, unsigned int num_entries)
{
  __shared__ float B_L2_NORM_SHARED;
  __shared__ float B[300];
  __shared__ float AB[16][32];


  int curr_entry = blockDim.y * blockIdx.x + threadIdx.y;
  // printf("%d\n",curr_entry);


  float A_L2_NORM = B_L2_NORM[curr_entry];
  float A_LOCAL[10];

  unsigned int Local_Count = 0;
  



  //Load the current entry
  if(curr_entry < num_entries){

    for(int i = 0; i < VECTOR_DIM;i += PARTITION_DIM){

      if(i+threadIdx.x < VECTOR_DIM){
        A_LOCAL[i / PARTITION_DIM] = A[VECTOR_DIM * curr_entry+i+threadIdx.x];
      }

    }
  }


      __syncthreads();

  //loop through all the entries
  for(unsigned int entry = 0 ; entry < num_entries; entry += 1){
  
    AB[threadIdx.y][threadIdx.x] = 0;

    if (entry < num_entries){

      if(threadIdx.y == 0 && threadIdx.x == 0){
          B_L2_NORM_SHARED = B_L2_NORM[entry];
      }

      int B_index = threadIdx.y * BLOCK_DIM_X + threadIdx.x;
      if( B_index < VECTOR_DIM){
        B[B_index] = A[entry * VECTOR_DIM + B_index];


      }
      __syncthreads();

      for(unsigned int partition = 0; partition < VECTOR_DIM; partition += PARTITION_DIM){
        
        if(partition + threadIdx.x < VECTOR_DIM){
          AB[threadIdx.y][threadIdx.x] += A_LOCAL[partition/PARTITION_DIM] * B[partition + threadIdx.x];
        }
        
      }

      __syncthreads();
      if(threadIdx.x < 16){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+16];
      }
      __syncthreads();
      if(threadIdx.x < 8){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+8];
      }
      __syncthreads();
      if(threadIdx.x < 4){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+4];
      }
      __syncthreads();
      if(threadIdx.x < 2){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+2];
      }
      __syncthreads();
      if(threadIdx.x < 1){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+1];
      }
      __syncthreads();


      if (threadIdx.x == 0 and curr_entry < num_entries){
        // printf("curr_entry %d,%d, %f\n", curr_entry,entry,AB[threadIdx.y][0] /(A_L2_NORM * B_L2_NORM_SHARED));
        if( AB[threadIdx.y][0] / (A_L2_NORM * B_L2_NORM_SHARED) > 0){
          Local_Count += 1;
        }
      }

    }
  }    
  if (threadIdx.x == 0 and curr_entry < num_entries){
    Count[curr_entry] = Local_Count-1;
  }


}




__global__ void 
cos_distance2(float *A, float* B, float *B_L2_NORM, int* Count, unsigned int offset, unsigned int num_entries)
{
  __shared__ float W_2[300];
  __shared__ float W_1[300];
  __shared__ float W_4_W_2[16][32];
  __shared__ float W_4_W_1[16][32];


  int curr_entry = blockDim.y * blockIdx.x + threadIdx.y + offset;

  float A_L2_NORM = B_L2_NORM[curr_entry];
  float W_4_LOCAL[10];

  unsigned int Local_Count = 0;
  



  //Load the current entry
  if(curr_entry < num_entries){

    for(int i = 0; i < VECTOR_DIM;i += PARTITION_DIM){

      if(i+threadIdx.x < VECTOR_DIM){
        W_4_LOCAL[i / PARTITION_DIM] = B[VECTOR_DIM * curr_entry+i+threadIdx.x];
      }

    }
  }


      __syncthreads();

  //loop through all the entries
  for(unsigned int entry = 0 ; entry < num_entries; entry += 1){
  
    W_4_W_2[threadIdx.y][threadIdx.x] = 0;
    W_4_W_1[threadIdx.y][threadIdx.x] = 0;

    if (entry < num_entries){


      int B_index = threadIdx.y * BLOCK_DIM_X + threadIdx.x;
      if( B_index < VECTOR_DIM){
        W_1[B_index] = A[entry * VECTOR_DIM + B_index];
        W_2[B_index] = B[entry * VECTOR_DIM + B_index];


      }
      __syncthreads();

      for(unsigned int partition = 0; partition < VECTOR_DIM; partition += PARTITION_DIM){
        
        if(partition + threadIdx.x < VECTOR_DIM){
          W_4_W_2[threadIdx.y][threadIdx.x] += W_4_LOCAL[partition/PARTITION_DIM] * W_2[partition + threadIdx.x];
          W_4_W_1[threadIdx.y][threadIdx.x] += W_4_LOCAL[partition/PARTITION_DIM] * W_1[partition + threadIdx.x];
        }
        
      }

      __syncthreads();
      if(threadIdx.x < 16){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+16];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+16];
      }
      __syncthreads();
      if(threadIdx.x < 8){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+8];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+8];
      }
      __syncthreads();
      if(threadIdx.x < 4){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+4];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+4];
      }
      __syncthreads();
      if(threadIdx.x < 2){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+2];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+2];
      }
      __syncthreads();
      if(threadIdx.x < 1){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+1];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+1];
      }
      __syncthreads();


      if (threadIdx.x == 0 and curr_entry < num_entries){
        // printf("curr_entry %d, %f\n", curr_entry,W_4_W_2[threadIdx.y][0] * A_L2_NORM / (W_4_W_2[threadIdx.y][0]) );
        if( W_4_W_2[threadIdx.y][0] * A_L2_NORM / (W_4_W_1[threadIdx.y][0]) > .50){
          Local_Count += 1;
        }
      }

    }
  }    
  if (threadIdx.x == 0 and curr_entry < num_entries){
    Count[curr_entry] = Local_Count;
  }


}
