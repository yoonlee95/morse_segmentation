
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_DIM_X 32 
#define BLOCK_DIM_Y 16 

#define VECTOR_DIM 300
#define PARTITION_DIM 32

__global__ void
vectorAdd(float *A, const float *B,unsigned int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = B[i] - A[i];
    }
}


__global__ void 
L2_norm(float *g_idata, float *g_odata)
{
  __shared__ float sdata[256];

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = g_idata[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * A; 

  if(tid + blockDim.x < 300){
    A = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * A;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   g_odata[blockIdx.x] = sqrtf(sdata[0]);
  }

}
__global__ void 
Dot_product(float *A_input, float *B_input,float * output)
{
  __shared__ float sdata[256];

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = A_input[tid + VECTOR_DIM * blockIdx.x];
  float B = B_input[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * B; 

  if(tid + blockDim.x < 300){
    A = A_input[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    A = B_input[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * B;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   output[blockIdx.x] = (sdata[0]);
  }

}



__global__ void 
Normalize_vector(float *g_idata)
{
  __shared__ float sdata[256];
  __shared__ float norm;

  unsigned int tid = threadIdx.x;

  sdata[tid] = 0;
  float A = g_idata[tid + VECTOR_DIM * blockIdx.x];

  sdata[tid] += A * A; 

  if(tid + blockDim.x < 300){
    A = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x];
    sdata[tid] += A * A;
  }

  __syncthreads();

  if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

  if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
  }
  __syncthreads();
  if(tid < 16){
    sdata[tid] += sdata[tid + 16];
  }
  __syncthreads();
  if(tid < 8){
    sdata[tid] += sdata[tid + 8];
  }
  __syncthreads();
  if(tid < 4){
    sdata[tid] += sdata[tid + 4];
  }
  __syncthreads();
  if(tid < 2){
    sdata[tid] += sdata[tid + 2];
  }
  __syncthreads();
  if(tid < 1){
    sdata[tid] += sdata[tid + 1];
  }
  __syncthreads();

  if (tid == 0){
    // printf("%f",sdata[0]);
   norm = sqrtf(sdata[0]);
  }
  __syncthreads();

  g_idata[tid + VECTOR_DIM * blockIdx.x] = g_idata[tid + VECTOR_DIM * blockIdx.x]/norm;

  if(tid + blockDim.x < 300){
    g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x] = g_idata[tid + blockDim.x + VECTOR_DIM * blockIdx.x]/ norm;
  }

}


__global__ void 
cos_distance(float *A, float *B_L2_NORM, int* Count, unsigned int num_entries)
{
  __shared__ float B_L2_NORM_SHARED;
  __shared__ float B[300];
  __shared__ float AB[16][32];


  int curr_entry = blockDim.y * blockIdx.x + threadIdx.y;
  // printf("%d\n",curr_entry);


  float A_L2_NORM = B_L2_NORM[curr_entry];
  float A_LOCAL[10];

  unsigned int Local_Count = 0;
  



  //Load the current entry
  if(curr_entry < num_entries){

    for(int i = 0; i < VECTOR_DIM;i += PARTITION_DIM){

      if(i+threadIdx.x < VECTOR_DIM){
        A_LOCAL[i / PARTITION_DIM] = A[VECTOR_DIM * curr_entry+i+threadIdx.x];
      }

    }
  }


      __syncthreads();

  //loop through all the entries
  for(unsigned int entry = 0 ; entry < num_entries; entry += 1){
  
    AB[threadIdx.y][threadIdx.x] = 0;

    if (entry < num_entries){

      if(threadIdx.y == 0 && threadIdx.x == 0){
          B_L2_NORM_SHARED = B_L2_NORM[entry];
      }

      int B_index = threadIdx.y * BLOCK_DIM_X + threadIdx.x;
      if( B_index < VECTOR_DIM){
        B[B_index] = A[entry * VECTOR_DIM + B_index];


      }
      __syncthreads();

      for(unsigned int partition = 0; partition < VECTOR_DIM; partition += PARTITION_DIM){
        
        if(partition + threadIdx.x < VECTOR_DIM){
          AB[threadIdx.y][threadIdx.x] += A_LOCAL[partition/PARTITION_DIM] * B[partition + threadIdx.x];
        }
        
      }

      __syncthreads();
      if(threadIdx.x < 16){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+16];
      }
      __syncthreads();
      if(threadIdx.x < 8){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+8];
      }
      __syncthreads();
      if(threadIdx.x < 4){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+4];
      }
      __syncthreads();
      if(threadIdx.x < 2){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+2];
      }
      __syncthreads();
      if(threadIdx.x < 1){
        AB[threadIdx.y][threadIdx.x] += AB[threadIdx.y][threadIdx.x+1];
      }
      __syncthreads();


      if (threadIdx.x == 0 and curr_entry < num_entries){
        // printf("curr_entry %d,%d, %f\n", curr_entry,entry,AB[threadIdx.y][0] /(A_L2_NORM * B_L2_NORM_SHARED));
        if( AB[threadIdx.y][0] / (A_L2_NORM * B_L2_NORM_SHARED) > 0){
          Local_Count += 1;
        }
      }

    }
  }    
  if (threadIdx.x == 0 and curr_entry < num_entries){
    Count[curr_entry] = Local_Count-1;
  }


}




__global__ void 
cos_distance2(float *A, float* B, float *B_L2_NORM, int* Count, unsigned int num_entries)
{
  __shared__ float W_2[300];
  __shared__ float W_1[300];
  __shared__ float W_4_W_2[16][32];
  __shared__ float W_4_W_1[16][32];


  int curr_entry = blockDim.y * blockIdx.x + threadIdx.y;
  // printf("%d\n",curr_entry);


  float A_L2_NORM = B_L2_NORM[curr_entry];
  float W_4_LOCAL[10];

  unsigned int Local_Count = 0;
  



  //Load the current entry
  if(curr_entry < num_entries){

    for(int i = 0; i < VECTOR_DIM;i += PARTITION_DIM){

      if(i+threadIdx.x < VECTOR_DIM){
        W_4_LOCAL[i / PARTITION_DIM] = B[VECTOR_DIM * curr_entry+i+threadIdx.x];
      }

    }
  }


      __syncthreads();

  //loop through all the entries
  for(unsigned int entry = 0 ; entry < num_entries; entry += 1){
  
    W_4_W_2[threadIdx.y][threadIdx.x] = 0;
    W_4_W_1[threadIdx.y][threadIdx.x] = 0;

    if (entry < num_entries){


      int B_index = threadIdx.y * BLOCK_DIM_X + threadIdx.x;
      if( B_index < VECTOR_DIM){
        W_1[B_index] = A[entry * VECTOR_DIM + B_index];
        W_2[B_index] = B[entry * VECTOR_DIM + B_index];


      }
      __syncthreads();

      for(unsigned int partition = 0; partition < VECTOR_DIM; partition += PARTITION_DIM){
        
        if(partition + threadIdx.x < VECTOR_DIM){
          W_4_W_2[threadIdx.y][threadIdx.x] += W_4_LOCAL[partition/PARTITION_DIM] * W_2[partition + threadIdx.x];
          W_4_W_1[threadIdx.y][threadIdx.x] += W_4_LOCAL[partition/PARTITION_DIM] * W_1[partition + threadIdx.x];
        }
        
      }

      __syncthreads();
      if(threadIdx.x < 16){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+16];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+16];
      }
      __syncthreads();
      if(threadIdx.x < 8){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+8];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+8];
      }
      __syncthreads();
      if(threadIdx.x < 4){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+4];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+4];
      }
      __syncthreads();
      if(threadIdx.x < 2){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+2];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+2];
      }
      __syncthreads();
      if(threadIdx.x < 1){
        W_4_W_2[threadIdx.y][threadIdx.x] += W_4_W_2[threadIdx.y][threadIdx.x+1];
        W_4_W_1[threadIdx.y][threadIdx.x] += W_4_W_1[threadIdx.y][threadIdx.x+1];
      }
      __syncthreads();


      if (threadIdx.x == 0 and curr_entry < num_entries){
        // printf("curr_entry %d, %f\n", curr_entry,W_4_W_2[threadIdx.y][0] * A_L2_NORM / (W_4_W_2[threadIdx.y][0]) );
        if( W_4_W_2[threadIdx.y][0] * A_L2_NORM / (W_4_W_1[threadIdx.y][0]) > .50){
          Local_Count += 1;
        }
      }

    }
  }    
  if (threadIdx.x == 0 and curr_entry < num_entries){
    Count[curr_entry] = Local_Count;
  }


}






// __global__ void vecAdd(float *w, float *out, int entry) {

//   __shared__ float w_curr_1_partition[BLOCK_DIM_Y][BLOCK_DIM_X];
//   __shared__ float w_curr_2_partition[BLOCK_DIM_Y][BLOCK_DIM_X];

//   float w_4_partition[10];
//   float w_3_partition[10];

//   float AB_partition[BLOCK_DIM_Y];
//   float B_partition[BLOCK_DIM_Y];
//   int cos_count = 0 ;

//   int index = blockIdx.x * (blockdim.y) + threadIdx.y;

//   int partition_index = 0;

//   int BLOCK_START_INDEX = index * 2 * VECTOR_DIM;
//   // Fill the W_4 and W_3 
//   for(int partition_index = 0; partition_index * PARTITION_DIM < VECTOR_DIM; partition_index += 1 )

//     if(partition_index * PARTITION_DIM< VECTOR_DIM){
//       w_4_partition[partition_index] = w[BLOCK_START_INDEX + VECTOR_DIM +  partition_index * PARTITION_DIM + threadIdx.x];
//       w_3_partition[partition_index] = w[BLOCK_START_INDEX + partition_index * PARTITION_DIM + threadIdx.x];
//     }
//   __syncthreads();
      

//   //Loop through the entries
//   for(int entry_index = 0; entry_index < entry; entry_index = entry_index + BLOCK_DIM_Y){




//       SHARED_START_INDEX= (entry_index + threadIdx.y) * 2 * VECTOR_DIM;
//       for(int shared_index = 0; shared_index < VECTOR_DIM; shared_index += PARTITION_DIM ){
//         w_curr_1_partition[threadIdx.y][threadIdx.x] = w[SHARED_START_INDEX + shared_index + threaIdx.x];
//         w_curr_2_partition[threadIdx.y][threadIdx.x] = w[SHARED_START_INDEX + VECTOR_DIM + shared_index + threadIdx.x];

//         __syncthreads();
//         int w_index = shared_index/PARTITION_DIM;


//         B = w_3_partition[w_index] - w_curr_2_partition[threadIdx.y][threadIdx.x] + w_curr_1_partition[threadIdx.y][threadIdx.x];
//         B_partition[threadIdx.y] += B * B;
//         AB_partition[threadIdx.y] += w_4_partition[w_index] * B 

//       }
      


//   }






//   // for(int i = index; i == index-1; i = (i+BLOCK_SIZE)%entry){
//   // }
//   // for(int i = index; i == index-1; i = (i+BLOCK_SIZE)%entry){

//   // }

//   // double cur_word;

//   // if (i < len){
//     out[i] = w[i+300*2] + w[i+300*1] - w[i];
//   // } 


// }